#include "hip/hip_runtime.h"
#include "./VectorAddSubtractTest.hpp"

#include "Hardware/GPU.hpp"
#include "Testing/TestResults.hpp"

namespace Testing {
	namespace Tests {
		__global__ void vectorAdd(const int* A, const int* B, int* C, int N) {
			int i = blockDim.x * blockIdx.x + threadIdx.x;
			if(i < N)
				C[i] = A[i] + B[i];
		}

		__global__ void vectorSubtract(const int* A, const int* B, int* C, int N) {
			int i = blockDim.x * blockIdx.x + threadIdx.x;
			if(i < N)
				C[i] = A[i] - B[i];
		}

		void VectorAddSubtractTest::doPass(hipStream_t stream) const {
			// Configuration
			size_t size = computeCount_ * sizeof(int);
			int threadsPerBlock = 256;
			int blocksPerGrid = (computeCount_ + threadsPerBlock - 1) / threadsPerBlock;

			// Allocate input vectors h_A and h_B in host memory
			// Don't bother to initialize
			int* hostVectorA = (int*) malloc(size);
			int* hostVectorB = (int*) malloc(size);
			int* hostVectorC = (int*) malloc(size);
			int* deviceVectorA;
			int* deviceVectorB;
			int* deviceVectorC;

			// Allocate vectors in device memory
			HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorA, size));
			HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorB, size));
			HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorC, size));

			HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(deviceVectorA, hostVectorA, size, hipMemcpyHostToDevice, stream));
			HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(deviceVectorB, hostVectorB, size, hipMemcpyHostToDevice, stream));

			// Run the kernels
			vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(deviceVectorA, deviceVectorB, deviceVectorC, computeCount_);
			vectorSubtract<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(deviceVectorA, deviceVectorB, deviceVectorC, computeCount_);

			HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(hostVectorC, deviceVectorC, size, hipMemcpyDeviceToHost, stream));

			if(stream == 0) {
				HARDWARE_GPU_HANDLE_API_CALL(hipDeviceSynchronize());
			} else {
				HARDWARE_GPU_HANDLE_API_CALL(hipStreamSynchronize(stream));
			}

			free(hostVectorA);
			free(hostVectorB);
			free(hostVectorC);
			hipFree(deviceVectorA);
			hipFree(deviceVectorB);
			hipFree(deviceVectorC);
		}

		TestResults VectorAddSubtractTest::onRun() {
			int devCount = 0;
			HARDWARE_GPU_HANDLE_API_CALL(hipGetDeviceCount(&devCount));
			for(int deviceNum = 0; deviceNum < devCount; deviceNum++) {
				hipDevice_t device;
				HARDWARE_GPU_HANDLE_API_CALL(hipDeviceGet(&device, deviceNum));

				char deviceName[32];
				HARDWARE_GPU_HANDLE_API_CALL(hipDeviceGetName(deviceName, 32, device));
				printf("Device Name: %s\n", deviceName);

				HARDWARE_GPU_HANDLE_API_CALL(hipSetDevice(deviceNum));
				// Do pass default stream
				doPass(0);

				// Do pass with user stream
				hipStream_t stream0;
				HARDWARE_GPU_HANDLE_API_CALL(hipStreamCreate(&stream0));
				doPass(stream0);

				hipDeviceSynchronize();

				// Flush all remaining CUPTI buffers before resetting the device.
				// This can also be called in the hipDeviceReset callback.
				cuptiActivityFlushAll(0);

				hipDeviceReset();
			}

			return TestResults(*this, {});
		}

		VectorAddSubtractTest::VectorAddSubtractTest(const int& computeCount)
			: Test("VectorAddSubtractTest")
			, computeCount_(computeCount) {
		}
	}
}