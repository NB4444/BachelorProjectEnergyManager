#include "hip/hip_runtime.h"
#include "./VectorAddSubtractTest.hpp"

#include "EnergyManager/Hardware/GPU.hpp"
#include "EnergyManager/Monitoring/GPUMonitor.hpp"
#include "EnergyManager/Testing/TestResults.hpp"

namespace EnergyManager {
	namespace Testing {
		namespace Tests {
			__global__ void vectorAdd(const int* A, const int* B, int* C, int N) {
				int i = blockDim.x * blockIdx.x + threadIdx.x;
				if(i < N) {
					C[i] = A[i] + B[i];
				}
			}

			__global__ void vectorSubtract(const int* A, const int* B, int* C, int N) {
				int i = blockDim.x * blockIdx.x + threadIdx.x;
				if(i < N) {
					C[i] = A[i] - B[i];
				}
			}

			void VectorAddSubtractTest::doPass(hipStream_t stream) const {
				// Configuration
				size_t size = computeCount_ * sizeof(int);
				int threadsPerBlock = 256;
				int blocksPerGrid = (computeCount_ + threadsPerBlock - 1) / threadsPerBlock;

				// Allocate input vectors h_A and h_B in host memory
				// Don't bother to initialize
				int* hostVectorA = (int*) malloc(size);
				int* hostVectorB = (int*) malloc(size);
				int* hostVectorC = (int*) malloc(size);
				int* deviceVectorA;
				int* deviceVectorB;
				int* deviceVectorC;

				// Allocate vectors in device memory
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorA, size));
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorB, size));
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMalloc((void**) &deviceVectorC, size));

				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(deviceVectorA, hostVectorA, size, hipMemcpyHostToDevice, stream));
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(deviceVectorB, hostVectorB, size, hipMemcpyHostToDevice, stream));

				// Run the kernels
				vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(deviceVectorA, deviceVectorB, deviceVectorC, computeCount_);
				vectorSubtract<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(deviceVectorA, deviceVectorB, deviceVectorC, computeCount_);

				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipMemcpyAsync(hostVectorC, deviceVectorC, size, hipMemcpyDeviceToHost, stream));

				if(stream == 0) {
					ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipDeviceSynchronize());
				} else {
					ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipStreamSynchronize(stream));
				}

				free(hostVectorA);
				free(hostVectorB);
				free(hostVectorC);
				hipFree(deviceVectorA);
				hipFree(deviceVectorB);
				hipFree(deviceVectorC);
			}

			std::map<std::string, std::string> VectorAddSubtractTest::onRun() {
				int devCount = 0;
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipGetDeviceCount(&devCount));

				hipDevice_t device;
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipDeviceGet(&device, gpu_->getID()));

				char deviceName[32];
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipDeviceGetName(deviceName, 32, device));
				printf("Device Name: %s\n", deviceName);

				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipSetDevice(gpu_->getID()));
				// Do pass default stream
				doPass(0);

				// Do pass with user stream
				hipStream_t stream0;
				ENERGY_MANAGER_HARDWARE_GPU_HANDLE_API_CALL(hipStreamCreate(&stream0));
				doPass(stream0);

				hipDeviceSynchronize();

				// Flush all remaining CUPTI buffers before resetting the device.
				// This can also be called in the hipDeviceReset callback.
				cuptiActivityFlushAll(0);

				hipDeviceReset();

				return {};
			}
			VectorAddSubtractTest::VectorAddSubtractTest(
				const std::string& name,
				const std::shared_ptr<Hardware::GPU>& gpu,
				const unsigned int& computeCount,
				std::map<std::shared_ptr<Monitoring::Monitor>, std::chrono::system_clock::duration> monitors)
				: Test(name, monitors)
				, gpu_(gpu)
				, computeCount_(computeCount) {
			}
		}
	}
}